
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    std::cout << "Hello, World!" << std::endl;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 0-th device
    std::cout << "num SM: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    std::cout << "Shared memory per block: " << deviceProp.sharedMemPerBlock << std::endl;
    return 0;
}
