#include <iostream>


#include "LbpUtils.h"
#include "Lbp.cu"

int main() {
//    std::cout << "Hello, World!" << std::endl;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, 0); // 0-th device
//    std::cout << "Num SM: " << deviceProp.multiProcessorCount << std::endl;
//    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
//    std::cout << "Shared memory per block: " << deviceProp.sharedMemPerBlock << std::endl;

    std::string csv_filename = "res/csv_images/leopard.csv";
    std::vector<std::vector<int>> vec = getImageFromCsv(csv_filename);

//    image as vector of vector of int to 2d linearized matrix
    int rows = vec.size();
    int cols = vec[0].size();
    int image[rows * cols];  // store image as a 2d linearized matrix
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++){
            image[i* cols + j] = vec[i][j];
//            std::cout << "vec: " << vec[i][j] << ", image: " << image[i][j] << std::endl;
        }
    }
    int new_image[rows * cols];
    lbpKernel<<<1,10>>>(image, new_image, cols, rows);


    return 0;

}